#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> 
#include "IntList.h"
#include "Pick.h"

__global__ 
void sliceCopyKernel(int *slice_out, int *slice_in, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N)
    {
        slice_out[idx] = slice_in[idx];
    }
}

__global__ 
void shaffleKernel(float *dst, float *src, int *indexdst, int nd, int N,  int *indexsrc, int *shapedst,    int *shapesrc, int *sizedst, int *sizesrc, int *pick)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    using namespace IntList;
    using namespace Pick;
    if (idx < N)
    {
        int ip = idx*nd;
        PositionToKernelIndex(sizesrc,idx,&indexsrc[ip]);
        ToPickInKernel(pick, &indexsrc[ip], &indexdst[ip]);
        int ps = KernelIndexPosition(shapedst, &indexdst[ip]);
        dst[ps] = src[idx];
    }

}

__global__ 
void ScatterKernel(int * shape, 
    int* indextensor, int N, int * tr, 
    float *u, float *tensor)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    using namespace IntList;
    using namespace Pick;   
   
    int nd = shape[0];
    if (idx < N)
    {
        int ip = idx*nd;
        for(int i = 0;i<nd;i++)
        {
            indextensor[ip+i] = tr[ip+i];
        }

        int ps = KernelIndexPosition(shape, &indextensor[ip]);
        tensor[ps] = u[idx];
    }
    
}


__global__ 
void ScatterXKernel(int * shape, 
    int* indextensor, int N, int * tr, 
    float *u, float *tensor)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    using namespace IntList;
    using namespace Pick;   
   
    int nd = shape[0];
    if (idx < N)
    {
        int ip = idx*nd;
        for(int i = 0;i<nd;i++)
        {
            indextensor[ip+i] = tr[ip+i];
        }

        int ps = KernelIndexPosition(shape, &indextensor[ip]);
        tensor[ps] = u[idx];
    }
    
}



